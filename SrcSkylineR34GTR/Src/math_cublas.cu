#include "math_cublas.hpp"
#include "benchmarking.hpp"
#include <hip/hip_runtime.h>
#include <mma.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <hip/hip_fp16.h>
#include <hipblas.h>
using namespace nvcuda;

MMAOptCublas::MMAOptCublas(float *A, float *B, float *C, float *Out, unsigned int size) :
        __matrixSize(size),
        _A(A),
        _B(B),
        _C(C),
        _Out(Out)
{
    hipblasHandle_t cublasHandle;
    hipblasCreate(&cublasHandle);
    cublasHandlerPtr = &cublasHandle;
}

const char *MMAOptCublas::GetOPTMame()
{
    return name;
}

double MMAOptCublas::Import()
{
    double time;
    BENCH_STORE(
    half *t_a = (half *)malloc(sizeof(half) * __matrixSize * __matrixSize);
    for (size_t i = 0; i < __matrixSize * __matrixSize; i++)
    {
        t_a[i] = (half)this->_A[i];
    }
    half *t_b = (half *)malloc(sizeof(half) * __matrixSize * __matrixSize);
    for (size_t i = 0; i < __matrixSize * __matrixSize; i++)
    {
        t_b[i] = (half)this->_B[i];
    }

    if (d_a == nullptr) hipMalloc((void **)&(this->d_a), sizeof(half) * __matrixSize*__matrixSize);
    hipMemcpy(this->d_a, t_a, __matrixSize*__matrixSize*sizeof(half), hipMemcpyHostToDevice);

    if (d_b == nullptr) hipMalloc((void **)(&this->d_b), sizeof(half) * __matrixSize*__matrixSize);
    hipMemcpy(this->d_b, t_b, __matrixSize*__matrixSize*sizeof(half), hipMemcpyHostToDevice);

    if (d_c == nullptr) hipMalloc((void **)&(this->d_c), sizeof(float) * __matrixSize*__matrixSize);
    hipMemcpy(this->d_c, this->_C, __matrixSize*__matrixSize*sizeof(float), hipMemcpyHostToDevice);

    free(t_a);
    free(t_b);
    ,time)
    return time;
}

double MMAOptCublas::Compute()
{
    float time;

    hipblasHandle_t cublasHandle = *(hipblasHandle_t*)cublasHandlerPtr;


    float alpha = 1, beta = 1;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                __matrixSize, __matrixSize, __matrixSize,
                &alpha,
                d_a, HIP_R_16F, __matrixSize,
                d_b, HIP_R_16F, __matrixSize,
                &beta,
                d_c, HIP_R_32F, __matrixSize,
                HIP_R_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    return (double)time;
}

double MMAOptCublas::Export()
{
    double time;
    BENCH_STORE(
    hipMemcpy(this->_Out, this->d_c, this->__matrixSize*this->__matrixSize*sizeof(float), hipMemcpyDeviceToHost);
    ,time)
    return time;
}

void MMAOptCublas::ComputeNTime(unsigned int loopCount)
{


    hipblasHandle_t cublasHandle = *(hipblasHandle_t*)cublasHandlerPtr;
    float alpha = 1, beta = 1;

    for (unsigned int i = 0; i < loopCount; i++)
    {
        hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                __matrixSize, __matrixSize, __matrixSize,
                &alpha,
                d_a, HIP_R_16F, __matrixSize,
                d_b, HIP_R_16F, __matrixSize,
                &beta,
                d_c, HIP_R_32F, __matrixSize,
                HIP_R_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);    
        }
    
}

void MMAOptCublas::Cleanup()
{
    if (d_a != nullptr) hipFree(d_a);
    if (d_b != nullptr) hipFree(d_b);
    if (d_c != nullptr) hipFree(d_c);
    d_a = nullptr;
    d_b = nullptr;
    d_c = nullptr;
    d_out = nullptr;
}