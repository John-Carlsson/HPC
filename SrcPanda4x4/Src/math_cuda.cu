#include "hip/hip_runtime.h"
#include "math_cuda.hpp"
#include "benchmarking.hpp"

#include <hip/hip_runtime_api.h>
#include <mma.h>
#include <stdio.h>

using namespace nvcuda;

#define MAT_MUL_ADD_ITERATION 2

__global__ void vector_add(float *out, float *a, float *b, int n)
{
    unsigned int task_id = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int task_count = blockDim.x * gridDim.x;
    unsigned int task_space = n / task_count;

    for (unsigned int i = task_id * task_space; i < (task_id + 1) * task_space; i++)
    {
        out[i] = a[i] + b[i];
    }
}

__global__ void mat_mul(float *out, float *a, float *b, int n)
{
    unsigned int task_id = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int task_count = blockDim.x * gridDim.x;
    unsigned int task_space = n / task_count;

    unsigned int i, j, k, s = task_id * task_space, e = (task_id + 1) * task_space;
    for (i = s; i < e; i ++)
        for (k = 0; k < n; k ++)
            for (j = 0; j < n; j ++)
                out[i*n+j] += a[i*n+k] * b[k*n+j];
}

#if MAT_MUL_ADD_ITERATION == 1
__global__ void mat_mul_add(float *out, float *a, float *b, float *c, int n)
{   
    // The kernel global id
    unsigned int task_id = blockIdx.x * blockDim.x + threadIdx.x;

    // The amount of kernels
    unsigned int task_count = blockDim.x * gridDim.x;

    // The amount of work for a kernel
    unsigned int task_space = n / task_count;

    unsigned int i, j, k, s = task_id * task_space, e = (task_id + 1) * task_space;
    for (i = s; i < e; i ++)
        for (k = 0; k < n; k ++)
            for (j = 0; j < n; j ++)
                out[i*n+j] += a[i*n+k] * b[k*n+j];

    for (i = s; i < e; i++)
        for (j = 0; j < n; j++)
            out[i*n+j] += c[i*n+j];
}
#elif MAT_MUL_ADD_ITERATION ==2
template<typename In, typename Out>
__global__ void mat_mul_add_global_mem(Out *out, In *a, In *b, Out *c, int n)
{   
    // The kernel global id
    unsigned int task_idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int task_idy = blockIdx.y * blockDim.y + threadIdx.y;

    float value = 0;
    for (unsigned int k = 0; k < n; k ++){
        float el_a = a[task_idy * n + k];
        float el_b = b[task_idx + n * k];
        value += (Out)el_a * el_b;
    }

    out[task_idy*n+task_idx] = c[task_idy*n+task_idx] + value;
}

template<typename In, typename Out, unsigned int TILE_SIZE>
__global__ void mat_mul_add(Out *out, In *A, In *B, Out *c, int n)
{
    __shared__ In a_shared [TILE_SIZE][TILE_SIZE];
    __shared__ In b_shared [TILE_SIZE][TILE_SIZE];
    
    // The kernel/thread global id, row i of left matrix
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;


    Out sum_acc = row < n && col < n ? c[row*n + col] : 0;
    for (size_t tileNum = 0; tileNum < n/TILE_SIZE; tileNum++)
    {

        // j is the column index of the left matrix
        int j = tileNum*TILE_SIZE + threadIdx.x;
        int i = tileNum*TILE_SIZE + threadIdx.y;

        // load into shared memory, coalesced
        a_shared[threadIdx.y][threadIdx.x] = A[row*n + j];
        b_shared[threadIdx.y][threadIdx.x] = B[i*n + col];
        
        __syncthreads();

        for (int k = 0; k < TILE_SIZE; k++)
        {
            sum_acc += (Out)a_shared[threadIdx.y][k] * (Out)b_shared[k][threadIdx.x];
            
        }
    }

    out[row*n + col] = sum_acc;
    
}   

template<typename In, typename Out, unsigned int TILE_SIZE, unsigned int FACTOR>
__global__ void mat_mul_add_rect(Out *out, In *A, In *B, Out *c, int n)
{
    __shared__ In a_shared [FACTOR][TILE_SIZE][TILE_SIZE];
    __shared__ In b_shared [TILE_SIZE][TILE_SIZE];
    
    // The kernel/thread global id, row i of left matrix
    unsigned int row = blockIdx.y * blockDim.y * FACTOR + threadIdx.y;
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;

    Out accs[FACTOR];
    for (unsigned int subthread = 0; subthread < FACTOR; subthread++)
        accs[subthread] = c[(row + TILE_SIZE * subthread)*n + col];

    for (unsigned int tileNum = 0; tileNum < n/TILE_SIZE; tileNum++)
    {
        unsigned int BRow = tileNum*TILE_SIZE + threadIdx.y;
        unsigned int BCol = col;

        b_shared[threadIdx.y][threadIdx.x] = (BRow < n && BCol < n) ? B[BRow*n + BCol] : (In)0;
        for (unsigned int subthread = 0; subthread < FACTOR; subthread++)
        {
            unsigned int ARow = row + TILE_SIZE * subthread;
            unsigned int ACol = tileNum*TILE_SIZE + threadIdx.x;
            a_shared[subthread][threadIdx.y][threadIdx.x] = (ARow < n && ACol < n) ? A[ARow*n + ACol] : (In)0;
            //printf("%d %d %d %d %d\n",subthread, row, col, ARow, ACol);
        }
        __syncthreads();
        for (unsigned int subthread = 0; subthread < FACTOR; subthread++)
            for (int k = 0; k < TILE_SIZE; k++)
            {
                accs[subthread] += (Out)(a_shared[subthread][threadIdx.y][k] * b_shared[k][threadIdx.x]);
            }
    }
    for (unsigned int subthread = 0; subthread < FACTOR; subthread++)
        out[(row + TILE_SIZE * subthread)*n + col] = accs[subthread];


    /*
    const unsigned int ratio = TILE_SIZE_Y/TILE_SIZE;

    for (unsigned int tileNumYA = 0; tileNumYA < n/(TILE_SIZE * FACTOR); tileNumYA++)
    {
        for (unsigned int tileNum  = 0; tileNum < FACTOR; tileNum++)
        {
            unsigned int ARow = row + TILE_SIZE * tileNum;
            unsigned int ACol = (tileNumYA * ratio + tileNum)*TILE_SIZE + threadIdx.x;
            unsigned int BRow = (tileNumYA * ratio + tileNum)*TILE_SIZE + threadIdx.y;
            unsigned int BCol = col;

            a_shared[tileNum][threadIdx.y][threadIdx.x] = (ARow < n && ACol < n) ? A[ARow*n + ACol] : (In)0;
            if (!tileNum)
                b_shared[threadIdx.y][threadIdx.x] = (BRow < n && BCol < n) ? B[BRow*n + BCol] : (In)0;

            __syncthreads();
        }
        for (unsigned int tileNum = 0; tileNum < ratio; tileNum++)
            for (int k = 0; k < TILE_SIZE; k++)
            {
                accs[tileNum] += (Out)(a_shared[tileNum][threadIdx.y][k] * b_shared[k][threadIdx.x]);
                
            }
    }
    for (unsigned int tileNum = 0; tileNum < ratio; tileNum++)
        out[(row + TILE_SIZE * tileNum)*n + col] = accs[tileNum];
*/
}   
        
#endif


void vector_add_cuda(float *out, float *a, float *b, int n)
{
    float *d_a;
    hipMalloc((void **)&d_a, sizeof(float) * n);

    hipMemcpy(d_a, a, sizeof(float) * n, hipMemcpyHostToDevice);

    vector_add<<<128, 64>>>(out, d_a, b, n);

    hipFree(d_a);
}

void matrix_multiplication_cuda(float *out, float *a, float *b, float *c, int n)
{
    float *d_a, *d_b, *d_out;

    hipMalloc((void **)&d_a, sizeof(float) * n*n);
    hipMemcpy(d_a, a, n*n*sizeof(float), hipMemcpyHostToDevice);

    hipMalloc((void **)&d_b, sizeof(float) * n*n);
    hipMemcpy(d_b, b, n*n*sizeof(float), hipMemcpyHostToDevice);

    hipMalloc((void **)&d_out, sizeof(float) * n*n);
    hipMemcpy(d_out, out, n*n*sizeof(float), hipMemcpyHostToDevice);

    mat_mul<<<32, 32>>>(d_out, d_a, d_b, n);

    hipMemcpy(out, d_out, n*n*sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
}

void matrix_multiplication_add_cuda(float *out, float *a, float *b, float *c, int n)
{
    float *d_a, *d_b, *d_c, *d_out;

    hipMalloc((void **)&d_a, sizeof(float) * n*n);
    hipMemcpy(d_a, a, n*n*sizeof(float), hipMemcpyHostToDevice);

    hipMalloc((void **)&d_b, sizeof(float) * n*n);
    hipMemcpy(d_b, b, n*n*sizeof(float), hipMemcpyHostToDevice);

    hipMalloc((void **)&d_out, sizeof(float) * n*n);
    hipMemcpy(d_out, out, n*n*sizeof(float), hipMemcpyHostToDevice);

    hipMalloc((void **)&d_c, sizeof(float) * n*n);
    hipMemcpy(d_c, c, n*n*sizeof(float), hipMemcpyHostToDevice);


#if     MAT_MUL_ADD_ITERATION == 1
    mat_mul_add<<<32, 32>>>(d_out, d_a, d_b, d_c, n);
#elif   MAT_MUL_ADD_ITERATION == 2
    dim3 numblocks(n/32,n/32);
    dim3 thread_per_block(32,32);
    mat_mul_add<float, float, 32><<<numblocks, thread_per_block>>>(d_out, d_a, d_b, d_c, n);
#elif   MAT_MUL_ADD_ITERATION == 3
    dim3 gridDim, blockDim;
	// 16 warps in one block
	blockDim.x = 16;
    blockDim.y = 16;

    gridDim.x = (n/blockDim.x);
    gridDim.y = (n/blockDim.y);
    mat_mul_add<<<gridDim, blockDim>>>(d_out, d_a, d_b, d_c, n);
#endif
    hipMemcpy(out, d_out, n*n*sizeof(float), hipMemcpyDeviceToHost);
    

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipFree(d_out);
}

MMAOptCUDA::MMAOptCUDA(float *A, float *B, float *C, float *Out, unsigned int size) :
        __matrixSize(size),
        _A(A),
        _B(B),
        _C(C),
        _Out(Out)
{
}

double MMAOptCUDA::Import()
{
    double time;
    BENCH_STORE(
    if (d_a == nullptr) hipMalloc((void **)&(this->d_a), sizeof(float) * __matrixSize*__matrixSize);
    hipMemcpy(this->d_a, this->_A, __matrixSize*__matrixSize*sizeof(float), hipMemcpyHostToDevice);

    if (d_b == nullptr) hipMalloc((void **)(&this->d_b), sizeof(float) * __matrixSize*__matrixSize);
    hipMemcpy(this->d_b, this->_B, __matrixSize*__matrixSize*sizeof(float), hipMemcpyHostToDevice);

    if (d_out == nullptr) hipMalloc((void **)&(this->d_out), sizeof(float) * __matrixSize*__matrixSize);
    hipMemcpy(this->d_out, this->_Out, __matrixSize*__matrixSize*sizeof(float), hipMemcpyHostToDevice);

    if (d_c == nullptr) hipMalloc((void **)&(this->d_c), sizeof(float) * __matrixSize*__matrixSize);
    hipMemcpy(this->d_c, this->_C, __matrixSize*__matrixSize*sizeof(float), hipMemcpyHostToDevice);

    ,time)
    return time;
}

const char *MMAOptCUDA::GetOPTMame()
{
    return name;
}

double MMAOptCUDA::Compute()
{
    if(d_out == nullptr) return -1;

    float time;
    dim3 gridDim, blockDim;
	// 16 warps in one block
	blockDim.x = 32;
    blockDim.y = 32;

    gridDim.x = (this->__matrixSize/blockDim.x);
    gridDim.y = (this->__matrixSize/blockDim.y);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    mat_mul_add<float, float, 32><<<gridDim, blockDim>>>(this->d_out, this->d_a, this->d_b, this->d_c, this->__matrixSize);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    return (double)time;
}

double MMAOptCUDA::Export()
{
    double time;
    BENCH_STORE(
    hipMemcpy(this->_Out, this->d_out, this->__matrixSize*this->__matrixSize*sizeof(float), hipMemcpyDeviceToHost);
        ,time)
    return time;
}

void MMAOptCUDA::ComputeNTime(unsigned int loopCount)
{
    if(d_out == nullptr) return;

    dim3 gridDim, blockDim;
	// 16 warps in one block
	blockDim.x = 16;
    blockDim.y = 16;

    gridDim.x = (this->__matrixSize/blockDim.x);
    gridDim.y = (this->__matrixSize/blockDim.y);
    for (unsigned int i = 0; i < loopCount; i++)
    {
        mat_mul_add<float, float, 16><<<gridDim, blockDim>>>(this->d_out, this->d_a, this->d_b, this->d_c, this->__matrixSize);
    }
    
}

void MMAOptCUDA::Cleanup()
{
    if (d_a != nullptr) hipFree(d_a);
    if (d_b != nullptr) hipFree(d_b);
    if (d_c != nullptr) hipFree(d_c);
    if (d_out != nullptr) hipFree(d_out);
    d_a = nullptr;
    d_b = nullptr;
    d_c = nullptr;
    d_out = nullptr;
}

double MMAOptCUDAGlobMem::Compute()
{
    if(d_out == nullptr) return -1;

    float time;
    dim3 numblocks(__matrixSize/32,__matrixSize/32);
    dim3 thread_per_block(32,32);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    mat_mul_add_global_mem<float, float><<<numblocks, thread_per_block>>>(d_out, d_a, d_b, d_c, __matrixSize);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    return (double)time;
   
}

void MMAOptCUDAGlobMem::ComputeNTime(unsigned int loopCount)
{
    if(d_out == nullptr) return;

    dim3 numblocks(__matrixSize/32,__matrixSize/32);
    dim3 thread_per_block(32,32);
   
    for (unsigned int i = 0; i < loopCount; i++)
        mat_mul_add_global_mem<float, float><<<numblocks, thread_per_block>>>(d_out, d_a, d_b, d_c, __matrixSize);
    
    
}

const char *MMAOptCUDAGlobMem::GetOPTMame()
{
    return name;
}

MMAOptCUDAH::MMAOptCUDAH(float *A, float *B, float *C, float *Out, unsigned int size) :
        __matrixSize(size),
        _A(A),
        _B(B),
        _C(C),
        _Out(Out)
{
}

double MMAOptCUDAH::Import()
{
    double time;
    BENCH_STORE(
    half *t_a = (half *)malloc(sizeof(half) * __matrixSize * __matrixSize);
    for (size_t i = 0; i < __matrixSize * __matrixSize; i++)
    {
        t_a[i] = (half)this->_A[i];
    }
    half *t_b = (half *)malloc(sizeof(half) * __matrixSize * __matrixSize);
    for (size_t i = 0; i < __matrixSize * __matrixSize; i++)
    {
        t_b[i] = (half)this->_B[i];
    }

    if (d_a == nullptr) hipMalloc((void **)&(this->d_a), sizeof(half) * __matrixSize*__matrixSize);
    hipMemcpy(this->d_a, t_a, __matrixSize*__matrixSize*sizeof(half), hipMemcpyHostToDevice);

    if (d_b == nullptr) hipMalloc((void **)(&this->d_b), sizeof(half) * __matrixSize*__matrixSize);
    hipMemcpy(this->d_b, t_b, __matrixSize*__matrixSize*sizeof(half), hipMemcpyHostToDevice);

    if (d_out == nullptr) hipMalloc((void **)&(this->d_out), sizeof(float) * __matrixSize*__matrixSize);
    hipMemcpy(this->d_out, this->_Out, __matrixSize*__matrixSize*sizeof(float), hipMemcpyHostToDevice);

    if (d_c == nullptr) hipMalloc((void **)&(this->d_c), sizeof(float) * __matrixSize*__matrixSize);
    hipMemcpy(this->d_c, this->_C, __matrixSize*__matrixSize*sizeof(float), hipMemcpyHostToDevice);

    free(t_a);
    free(t_b);
    ,time)
    return time;
}

const char *MMAOptCUDAH::GetOPTMame()
{
    return name;
}

double MMAOptCUDAH::Compute()
{
    if(d_out == nullptr) return -1;

    float time;
    dim3 gridDim, blockDim;
	// 16 warps in one block
	blockDim.x = 32;
    blockDim.y = 32;

    gridDim.x = (this->__matrixSize/blockDim.x);
    gridDim.y = (this->__matrixSize/(blockDim.y*2));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    mat_mul_add_rect<half, float, 32, 2><<<gridDim, blockDim>>>(d_out, (half*)d_a, (half*)d_b, d_c, __matrixSize);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    return (double)time;
}

double MMAOptCUDAH::Export()
{
    double time;
    BENCH_STORE(
    hipMemcpy(this->_Out, this->d_out, this->__matrixSize*this->__matrixSize*sizeof(float), hipMemcpyDeviceToHost);
        ,time)
    return time;
}

void MMAOptCUDAH::ComputeNTime(unsigned int loopCount)
{
    if(d_out == nullptr) return;

    dim3 gridDim, blockDim;
	// 16 warps in one block
	blockDim.x = 16;
    blockDim.y = 16;

    gridDim.x = (this->__matrixSize/blockDim.x);
    gridDim.y = (this->__matrixSize/blockDim.y);
    for (unsigned int i = 0; i < loopCount; i++)
    {
        mat_mul_add<half, float, 16><<<gridDim, blockDim>>>(d_out, (half*)d_a, (half*)d_b, d_c, __matrixSize);
    }
    
}

void MMAOptCUDAH::Cleanup()
{
    if (d_a != nullptr) hipFree(d_a);
    if (d_b != nullptr) hipFree(d_b);
    if (d_c != nullptr) hipFree(d_c);
    if (d_out != nullptr) hipFree(d_out);
    d_a = nullptr;
    d_b = nullptr;
    d_c = nullptr;
    d_out = nullptr;
}


const char *MMAOptCUDASF16::GetOPTMame()
{
    return name;
}

double MMAOptCUDASF16::Compute()
{
    if(d_out == nullptr) return -1;

    float time;
    dim3 gridDim, blockDim;
	// 16 warps in one block
	blockDim.x = 16;
    blockDim.y = 16;

    gridDim.x = (this->__matrixSize/blockDim.x);
    gridDim.y = (this->__matrixSize/blockDim.y);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    mat_mul_add<float, float, 16><<<gridDim, blockDim>>>(this->d_out, this->d_a, this->d_b, this->d_c, this->__matrixSize);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    return (double)time;
}



const char *MMAOptCUDASH32::GetOPTMame()
{
    return name;
}

double MMAOptCUDASH32::Compute()
{
    if(d_out == nullptr) return -1;

    float time;
    dim3 gridDim, blockDim;
	// 16 warps in one block
	blockDim.x = 32;
    blockDim.y = 32;

    gridDim.x = (this->__matrixSize/blockDim.x);
    gridDim.y = (this->__matrixSize/blockDim.y);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    mat_mul_add<half, float, 32><<<gridDim, blockDim>>>(d_out, (half*)d_a, (half*)d_b, d_c, __matrixSize);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    return (double)time;
}

const char *MMAOptCUDAH2::GetOPTMame()
{
    return name;
}

double MMAOptCUDAH2::Compute()
{
    if(d_out == nullptr) return -1;

    float time;
    dim3 gridDim, blockDim;
	// 16 warps in one block
	blockDim.x = 32;
    blockDim.y = 32;

    gridDim.x = (this->__matrixSize/blockDim.x);
    gridDim.y = (this->__matrixSize/(blockDim.y*3));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    mat_mul_add_rect<half, float, 32, 3><<<gridDim, blockDim>>>(d_out, (half*)d_a, (half*)d_b, d_c, __matrixSize);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    return (double)time;
}